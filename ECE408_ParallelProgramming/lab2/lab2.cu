#include "hip/hip_runtime.h"
// LAB 2 FA24

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)


// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns)
{
  //@@ Implement matrix multiplication kernel here

  // Get row, column index; find k
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;

  // Do computing
  if( (row < numCRows) && (col < numCColumns) ){
    float pvalue = 0;
    for( int k = 0; k < numAColumns; k++ ){
      pvalue += A[row*numAColumns+k] * B[k*numBColumns+col]; //check index
    }
    // write back
    C[row*numCColumns+col] = pvalue;
  }
}


int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  //@@ Importing data and creating memory on host
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;

  //@@ Allocate the hostC matrix
  hostC = (float*)malloc(numCRows*numCColumns*sizeof(float));

  //@@ Allocate GPU memory here
  float *deviceA, *deviceB, *deviceC;
  hipMalloc((void**)&deviceA, numARows*numAColumns*sizeof(float));
  hipMalloc((void**)&deviceB, numBRows*numBColumns*sizeof(float));
  hipMalloc((void**)&deviceC, numCRows*numCColumns*sizeof(float));

  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows*numAColumns*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows*numBColumns*sizeof(float), hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(numCColumns/32.0),ceil(numCRows/32.0),1);
  dim3 DimBlock(32,32,1); // use the default warp size

  //@@ Launch the GPU Kernel here
  matrixMultiply<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  hipDeviceSynchronize();
  
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows*numCColumns*sizeof(float), hipMemcpyDeviceToHost);

  // debug
  // wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);

  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);


  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  //@@Free the hostC matrix
  free(hostC);

  return 0;
}

