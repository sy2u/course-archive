#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define TILE_WIDTH  8
#define MASK_WIDTH  3

//@@ Define constant memory for device kernel here
__constant__ float M_c[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;

  // use strategy 2
  int row_o = blockIdx.y * TILE_WIDTH + ty;
  int col_o = blockIdx.x * TILE_WIDTH + tx;
  int z_o = blockIdx.z * TILE_WIDTH + tz;

  int radius = (MASK_WIDTH-1)/2;
  int row_i = row_o - radius;
  int col_i = col_o - radius;
  int z_i = z_o - radius;

  // load data
  __shared__ float tile [TILE_WIDTH+(MASK_WIDTH-1)][TILE_WIDTH+(MASK_WIDTH-1)][TILE_WIDTH+(MASK_WIDTH-1)];

  if( (row_i>=0) && (row_i<y_size) && (col_i>=0) && (col_i<x_size) && (z_i>=0) && (z_i<z_size) ){
    tile[tz][ty][tx] = input[z_i*x_size*y_size + row_i*x_size + col_i];
  } else {
    tile[tz][ty][tx] = 0.0f;
  }

  __syncthreads();
  
  float pvalue = 0.0f;
  if((tx < TILE_WIDTH) && (ty < TILE_WIDTH) && (tz < TILE_WIDTH)){
    // do computation
    for( int i = 0; i < MASK_WIDTH; i++ ){
      for( int j = 0; j < MASK_WIDTH; j++ ){
        for( int k = 0; k < MASK_WIDTH; k++ ){
          pvalue += M_c[i][j][k] * tile[i+tz][j+ty][k+tx];
        }
      }
    }
    // write back
    if( (z_o<z_size) && (row_o<y_size) && (col_o<x_size) ){
      output[z_o*x_size*y_size + row_o*x_size + col_o] = pvalue;
    }
    
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  //@@ Initial deviceInput and deviceOutput here.
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);


  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first three elements were the dimensions
  hipMalloc((void**)&deviceInput, (inputLength-3)*sizeof(float));
  hipMalloc((void**)&deviceOutput, (inputLength-3)*sizeof(float));

  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do not need to be copied to the gpu
  hipMemcpy(deviceInput, hostInput+3, (inputLength-3)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(M_c), hostKernel, kernelLength*sizeof(float), 0, hipMemcpyHostToDevice);

  //@@ Initialize grid and block dimensions here
  dim3 DimGrid(ceil(x_size/(1.0*TILE_WIDTH)),ceil(y_size/(1.0*TILE_WIDTH)),ceil(z_size/(1.0*TILE_WIDTH)));
  dim3 DimBlock(TILE_WIDTH+(MASK_WIDTH-1),TILE_WIDTH+(MASK_WIDTH-1),TILE_WIDTH+(MASK_WIDTH-1));

  //@@ Launch the GPU kernel here
  conv3d<<<DimGrid,DimBlock>>>(deviceInput,deviceOutput,z_size,y_size,x_size);

  hipDeviceSynchronize();

  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(hostOutput+3, deviceOutput, (inputLength-3)*sizeof(float), hipMemcpyDeviceToHost);

  // debug
  // for(int i = 0; i < 10; i++){
  //   wbLog(TRACE, "hostOutput[", i, "] = ", hostOutput[i]);
  // }

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  //@@ Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
