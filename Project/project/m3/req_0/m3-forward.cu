#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define BLOCK_SIZE 256
#define STREAM_NUM 4

const float *host_input_temp;
const float *host_output_temp;

__global__ void matrix_unrolling_kernel(const float *input, float *output,
                                        const int Batch, const int Channel,
                                        const int Height, const int Width,
                                        const int K) {
    /*
    Modify this function to implement the input matrix unrolling kernel.

    Function paramter definitions:
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)

    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    
    // TODO: Insert your input matrix unrolling kernel code here
    // int Height_unrolled = K * K * Channel;
    int Width_unrolled = Batch * Height_out * Width_out;

    #define out_3d(i2, i1, i0) output[(i2) * (Height_out * Width_out) + (i1) * (Width_unrolled) + i0]

    // int H_grid = (Height_out + TILE_WIDTH - 1) / TILE_WIDTH;
    size_t W_grid = (Width_out + TILE_WIDTH - 1) / TILE_WIDTH;

    int b = blockIdx.z;
    int c = blockIdx.x;
    int h = (blockIdx.y / W_grid) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.y % W_grid) * TILE_WIDTH + threadIdx.x;

    if( h < Height_out && w < Width_out ){
        int w_base = c * (K*K);
        for( int p = 0; p < K; p++ ){
            for( int q = 0; q < K; q++ ){
                size_t h_unroll = w_base + p * K + q;
                size_t w_unroll = h * Width_out + w;
                out_3d(b, h_unroll, w_unroll) = in_4d(b ,c, h+p, w+q);
            }
        }
    } 

    #undef in_4d
    #undef out_3d
}

// Tiled matrix multiplication kernel. Computes C = AB
// You don't need to modify this kernel.
__global__ void matrixMultiplyShared(const float *A, const float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;
    float val = 0;

    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) {
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = A[(size_t) row * numAColumns + tileId * TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0;
        }
        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = B[((size_t) tileId * TILE_WIDTH + ty) * numBColumns + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            for (int i = 0; i < TILE_WIDTH; i++) {
                val += tileA[ty][i] * tileB[i][tx];
            }
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = val;
    }
}

// Permutes the matmul result.
// The output feature map after matmul is of shape Map_out x Batch x Height_out x Width_out,
// and we need to permute it into Batch x Map_out x Height_out x Width_out.
// You don't need to modify this kernel.
__global__ void matrix_permute_kernel(const float *input, float *output, int Map_out,
                                      int Batch, int image_size) {
    int b = blockIdx.y;
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (x < image_size) {
        for (int m = 0; m < Map_out; m++) {
            output[b * Map_out * image_size + m * image_size + x] =
                    input[m * Batch * image_size + b * image_size + x];
        }
    }
}


__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU
    // We pass double pointers for you to initialize the relevant device pointers,
    // which are passed to the other two functions.
    hipMalloc(device_input_ptr, (size_t) sizeof(float)*Batch*Channel*Height*Width);
    hipMalloc(device_mask_ptr, (size_t) sizeof(float)*K*K*Channel*Map_out);
    hipMalloc(device_output_ptr, (size_t) sizeof(float)*Batch*(Height-K+1)*(Width-K+1)*Map_out);

    // small dataset, no need to copy by stream
    hipMemcpy(*device_mask_ptr, host_mask,  (size_t) sizeof(float)*K*K*Channel*Map_out, hipMemcpyHostToDevice);

    // prepare for stream
    host_input_temp = host_input;
    host_output_temp = host_output;

    // Useful snippet for error checking
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}

__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const size_t Height_unrolled = Channel * K * K;
    
    float *unrolled_matrix;  // Pointer to device memory for storing the unrolled matrix
    float *matmul_output;    // Pointer to device memory for storing the result of matrix multiplication
    hipMalloc((void**)&unrolled_matrix, (size_t) Batch * Channel * K * K * Height_out * Width_out * sizeof(float));
    hipMalloc((void**)&matmul_output, (size_t) (Batch * Map_out * Height_out * Width_out) * sizeof(float));

    ///////////////////////////////
    // [PM3] Req_0: Stream Start //
    ///////////////////////////////

    hipStream_t stream [STREAM_NUM]; 

    for (int i = 0; i < STREAM_NUM; i++){

        hipError_t err = hipStreamCreate(&stream[i]);
        if (err != hipSuccess) {
            printf("Stream creation failed for stream[%d]: %s\n", i, hipGetErrorString(err));
            return;
        }

        size_t  batch_size = (Batch+STREAM_NUM-1) / STREAM_NUM;;
        size_t  offset_input = (size_t) Channel*Height*Width* batch_size;
        size_t  offset_unrolled = (size_t) Channel*K*K*Height_out*Width_out* batch_size;
        size_t  offset_output = (size_t) Height_out*Width_out*Map_out* batch_size;

        size_t  curr_batch = (i==STREAM_NUM-1) ? Batch-(STREAM_NUM-1)*batch_size : batch_size;
        size_t  Width_unrolled = curr_batch * Height_out * Width_out;

        // Copy: Host -> Device
        size_t  size_input = (size_t) Channel*Height*Width* curr_batch;
        hipMemcpyAsync((void*)(device_input+offset_input*i), (void*)(host_input_temp+offset_input*i), sizeof(float)*size_input, hipMemcpyHostToDevice, stream[i]);

        // Kernel Call: Unroll
        dim3 Grid_unroll(Channel, (size_t) (Height_out+TILE_WIDTH-1)/TILE_WIDTH * (Width_out+TILE_WIDTH-1)/TILE_WIDTH, curr_batch);
        dim3 Block_unroll(TILE_WIDTH, TILE_WIDTH, 1);
        matrix_unrolling_kernel<<<Grid_unroll, Block_unroll, 0, stream[i]>>>
            (device_input+offset_input*i, unrolled_matrix+offset_unrolled*i, curr_batch, Channel, Height, Width, K);

        // Kernel Call: Matmul
        dim3 Grid_matmul((size_t)(Width_unrolled+TILE_WIDTH-1)/TILE_WIDTH, (size_t)(Map_out+TILE_WIDTH-1)/TILE_WIDTH, 1);
        dim3 Block_matmul(TILE_WIDTH, TILE_WIDTH, 1);
        matrixMultiplyShared<<<Grid_matmul,Block_matmul, 0, stream[i]>>>
            (device_mask, unrolled_matrix+offset_unrolled*i, matmul_output+offset_output*i, Map_out, Height_unrolled, Height_unrolled, Width_unrolled, Map_out, Width_unrolled);

        // Kernel Call: Permute
        const int out_image_size = Height_out * Width_out;
        dim3 permute_kernel_grid_dim((out_image_size - 1) / BLOCK_SIZE + 1, curr_batch, 1);
        matrix_permute_kernel<<<permute_kernel_grid_dim, BLOCK_SIZE, 0, stream[i]>>>
            (matmul_output+offset_output*i, device_output+offset_output*i, Map_out, curr_batch, out_image_size);

        // Copy: Device -> Host
        size_t  size_output = (size_t) Height_out*Width_out*Map_out* curr_batch;
        hipMemcpyAsync((void*)(host_output_temp+offset_output*i), (void*)(device_output+offset_output*i), sizeof(float)*size_output, hipMemcpyDeviceToHost, stream[i]);
    }
    hipDeviceSynchronize();

    for (int i = 0; i < STREAM_NUM; i++) {
        hipStreamDestroy(stream[i]);
    }

    /////////////////////////////
    // [PM3] Req_0: Stream End //
    /////////////////////////////

    hipFree(matmul_output);
    hipFree(unrolled_matrix);
    
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Already done in conv_forward_gpu for Stream
    
    // const int Height_out = Height - K + 1;
    // const int Width_out = Width - K + 1;
    // hipMemcpy(host_output, device_output, (size_t) sizeof(float)*Height_out*Width_out*Map_out*Batch, hipMemcpyDeviceToHost);

    // TODO: Free device memory
    hipFree(device_input);
    hipFree(device_mask);
    hipFree(device_output);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}